// pinned memepry of 3 zeros
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
// time stamp function in seconds
#include <sys/time.h>
#include <math.h>
double getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

__global__ void f_siggen(float *d_X, float *d_Y, float *d_Z, int n, int m)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // n: n_row, m: n_col
    if (i < n && j < m)
    {
        float a = (i > 0) ? d_X[(i - 1) * m + j] : 0;
        float x = d_X[i * m + j];
        float b = (i < n - 1) ? d_X[(i + 1) * m + j] : 0;

        float c = (j > 1) ? d_Y[i * m + (j - 2)] : 0;
        float d = (j > 0) ? d_Y[i * m + (j - 1)] : 0;
        float y = d_Y[i * m + j];

        d_Z[i * m + j] = a + x + b - c - d - y;
    }
}

__host__ void matrixSumHost(float *h_X, float *h_Y, float *h_Z, int n_row, int n_col)
{
    for (int i = 0; i < n_row; i++)
    {
        for (int j = 0; j < n_col; j++)
        {
            float a = (i > 0) ? h_X[(i - 1) * n_col + j] : 0;
            float x = h_X[i * n_col + j];
            float b = (i < n_row - 1) ? h_X[(i + 1) * n_col + j] : 0;
            float c = (j > 1) ? h_Y[i * n_col + (j - 2)] : 0;
            float d = (j > 0) ? h_Y[i * n_col + (j - 1)] : 0;
            float y = h_Y[i * n_col + j];
            h_Z[i * n_col + j] = a + x + b - d - c - y;
        }
    }
}

__host__ void iniData(float *h_X, float *h_Y, int n_row, int n_col)
{
    for (int i = 0; i < n_row; i++)
    {
        for (int j = 0; j < n_col; j++)
        {
            h_X[i * n_col + j] = (float)((i + j) % 100) / 2.0;
            h_Y[i * n_col + j] = (float)3.25 * ((i + j) % 100);
        }
    }
}

__host__ bool checkEqual(float *h_Z, float *h_dZ, int n_row, int n_col)
{
    float epsilon = 1e-6; // Adjust based on acceptable tolerance
    for (int i = 0; i < n_row; i++)
    {
        for (int j = 0; j < n_col; j++)
        {
            // printf("%f ", h_dZ[i * n_col + j]);
            if (fabsf(h_Z[i * n_col + j] - h_dZ[i * n_col + j]) > epsilon)
            {
                return false;
            }
        }
        // printf("\n");
    }
    return true;
}

int main(int argc, char *argv[])
{

    // set matrix size
    int n_row = 0;
    int n_col = 0;
    if (argc != 3)
    {
        printf("Error: The number of arguments is not 2");
    }
    else
    {
        n_row = atoi(argv[1]);
        n_col = atoi(argv[2]);
    }
    int noElems = n_row * n_col;
    int bytes = noElems * sizeof(float);
    hipError_t status;
    dim3 block_size(32, 32);
    dim3 grid_size((n_col + block_size.x - 1) / block_size.x, (n_row + block_size.y - 1) / block_size.y);

    // alloc memory host-size
    float *h_X, *h_Y, *h_dZ;
    status = hipHostAlloc((void **)&h_X, bytes, 0);
    if (status != hipSuccess)
    {
        printf("Error: hipHostAlloc for h_X failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    status = hipHostAlloc((void **)&h_Y, bytes, 0);
    if (status != hipSuccess)
    {
        printf("Error: hipHostAlloc for h_Y failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    float *h_Z = (float *)malloc(bytes);
    status = hipHostAlloc((void **)&h_dZ, bytes, 0);
    if (status != hipSuccess)
    {
        printf("Error: hipHostAlloc for h_dZ failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    // initialization
    iniData(h_X, h_Y, n_row, n_col);

    // alloc memory dev-size
    float *d_X, *d_Y, *d_Z;
    status = hipMalloc((void **)&d_X, bytes);
    if (status != hipSuccess)
    {
        printf("Error: hipMalloc for h_X failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    status = hipMalloc((void **)&d_Y, bytes);
    if (status != hipSuccess)
    {
        printf("Error: hipMalloc for h_Y failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    status = hipMalloc((void **)&d_Z, bytes);
    if (status != hipSuccess)
    {
        printf("Error: hipMalloc for d_Z failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    // transfer data to dev
    double t_G_t_1 = getTimeStamp();
    double CG_t_t_1 = getTimeStamp();
    status = hipMemcpy(d_X, h_X, bytes, hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        printf("Error: hipMemcpy h_X -> d_X failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    status = hipMemcpy(d_Y, h_Y, bytes, hipMemcpyHostToDevice);
    if (status != hipSuccess)
    {
        printf("Error: hipMemcpy h_Y -> d_Y failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    double CG_t_t_2 = getTimeStamp();

    // invoke a kernel
    double k_t_1 = getTimeStamp();
    f_siggen<<<grid_size, block_size>>>(d_X, d_Y, d_Z, n_row, n_col);
    status = hipDeviceSynchronize();
    if (status != hipSuccess)
    {
        printf("Error: hipDeviceSynchronize failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    double k_t_2 = getTimeStamp();

    // copy data back
    double GC_t_t_1 = getTimeStamp();
    status = hipMemcpy(h_dZ, d_Z, bytes, hipMemcpyDeviceToHost);
    if (status != hipSuccess)
    {
        printf("Error: hipMemcpy d_Z -> h_dZ failed: %s\n", hipGetErrorString(status));
        return -1;
    }
    double GC_t_t_2 = getTimeStamp();
    double t_G_t_2 = getTimeStamp();

    // check result
    matrixSumHost(h_X, h_Y, h_Z, n_row, n_col);
    printf("the two matrices are equal: %d\n", checkEqual(h_Z, h_dZ, n_row, n_col));

    // output time and exit
    float k_t = k_t_2 - k_t_1;
    float t_G_t = t_G_t_2 - t_G_t_1;
    float CG_t_t = CG_t_t_2 - CG_t_t_1;
    float GC_t_t = GC_t_t_2 - GC_t_t_1;
    float Z_v = h_dZ[5 * n_col + 5];
    printf("%.6f %.6f %.6f %.6f %.6f\n", t_G_t, CG_t_t, k_t, GC_t_t, Z_v);

    // free memory resources
    // free memory resources
    hipFree(d_X);
    hipFree(d_Y);
    hipFree(d_Z);

    hipHostFree(h_X);
    hipHostFree(h_Y);
    free(h_Z);
    hipHostFree(h_dZ);
    hipDeviceReset();
    exit(EXIT_SUCCESS);
}
