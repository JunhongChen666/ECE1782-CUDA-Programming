/*
* ECE1782 - W2024 - Lab 2 - Sample Code
* Sample Test Cases (sum)
n, result 
100,18295201.010496
200,147100808.124588
300,497296827.464880
400,1179763265.153962
500,2305380127.308517
600,3985027420.060339
700,6329585154.758305
800,9449933335.045414
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <math.h>

#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code == hipSuccess)
        return;

    fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort)
        exit(code);
}

/*Use the following to get a timestamp*/
double getTimeStamp()
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_usec / 1000000 + tv.tv_sec;
}

void iniData(float *B, int n)
{
    int i, j, k;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            for (k = 0; k < n; k++)
            {
                B[i * n * n + j * n + k] = (float)((i + j + k) % 10) * (float)1.1;
            }
        }
    }
    // for (i = 0; i < n; i++)
    // {
    //     for (j = 0; j < n; j++)
    //     {

    //         for (k = 0; k < n; k++)
    //         {
    //             printf("B: %f", B[i * n * n + j * n + k]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
}

void computeOnCPU(float *A, float *B, int n)
{
    int i, j, k;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            for (k = 0; k < n; k++)
            {
                float a = (i - 1 >= 0) ? B[(i - 1) * n * n + j * n + k] : 0;
                float b = (i + 1 < n) ? B[(i + 1) * n * n + j * n + k] : 0;
                float c = (j - 1 >= 0) ? B[i * n * n + (j - 1) * n + k] : 0;
                float d = (j + 1 < n) ? B[i * n * n + (j + 1) * n + k] : 0;
                float e = (k - 1 >= 0) ? B[i * n * n + j * n + k - 1] : 0;
                float f = (k + 1 < n) ? B[i * n * n + j * n + k + 1] : 0;
                A[i * n * n + j * n + k] = (float)0.8 * (a + b + c + d + e + f);
            }
        }
    }
}

__global__ void kernel(float *A, float *B, int n)
{

    // int x = blockIdx.x * blockDim.x + threadIdx.x;
    // int y = blockIdx.y * blockDim.y + threadIdx.y;
    // int z = blockIdx.z * blockDim.z + threadIdx.z;
    // int idx = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
    int idx = blockDim.x * blockDim.y * blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x;
    // printf("idx %d\n", idx);
    int i = idx / (n * n);
    int z = idx % (n * n);
    int j = z / n;
    int k = z % n;
    if (i < n && j < n && k < n)
    {
        // printf("K: %d j: %d i: %d idx: %d\n", k, j, i, idx);
        float a = (i - 1 >= 0) ? B[(i - 1) * n * n + j * n + k] : 0;
        float b = (i + 1 < n) ? B[(i + 1) * n * n + j * n + k] : 0;
        float c = (j - 1 >= 0) ? B[i * n * n + (j - 1) * n + k] : 0;
        float d = (j + 1 < n) ? B[i * n * n + (j + 1) * n + k] : 0;
        float e = (k - 1 >= 0) ? B[i * n * n + j * n + k - 1] : 0;
        float f = (k + 1 < n) ? B[i * n * n + j * n + k + 1] : 0;
        A[i * n * n + j * n + k] = (float)0.8 * (a + b + c + d + e + f);
    }
}

bool checkEqual(float *h_A, float *h_dA, int n)
{
    float epsilon = 1e-6;
    int i, j, k;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            for (k = 0; k < n; k++)
            {
                // printf("da: %f ", h_dA[i * n * n + j * n + k]);
                if (fabsf(h_A[i * n * n + j * n + k] - h_dA[i * n * n + j * n + k]) > epsilon)
                {
                    printf("%d %d %d", i, j, k);
                    return false;
                }
            }
            // printf("\n");
        }
        // printf("\n");
    }
    return true;
}

double getSum(float *h_dA, int n)
{
    int i, j, k;
    double sum = 0;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            for (k = 0; k < n; k++)
            {
                sum += h_dA[i * n * n + j * n + k] * (((i + j + k) % 10) ? 1 : -1);
            }
        }
    }
    return sum;
}

int main(int argc, char *argv[])
{

    // set matrix size
    if (argc != 2)
    {
        printf("Error: wrong number of args\n");
        exit(1);
    }

    int n = atoi(argv[1]);
    size_t number_of_elements = ((size_t)n) * n * n;
    size_t bytes = number_of_elements * sizeof(float);
    gpuErrchk(hipDeviceReset());

    float *h_A, *h_B, *h_dA;
    gpuErrchk(hipHostAlloc((void **)&h_A, bytes, 0));
    gpuErrchk(hipHostAlloc((void **)&h_B, bytes, 0));
    gpuErrchk(hipHostAlloc((void **)&h_dA, bytes, 0));
    iniData(h_B, n);

    double start_time = getTimeStamp();
    float *d_A, *d_B;
    gpuErrchk(hipMalloc((void **)&d_A, bytes));
    gpuErrchk(hipMalloc((void **)&d_B, bytes));

    dim3 block_size(32, 32);
    dim3 grid_size((n * n * n + block_size.x * block_size.y - 1) / (block_size.x * block_size.y)); // 2 grids

    gpuErrchk(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    kernel<<<grid_size, block_size>>>(d_A, d_B, n);
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(h_dA, d_A, bytes, hipMemcpyDeviceToHost));
    double end_time = getTimeStamp();

    computeOnCPU(h_A, h_B, n);

    // int i, j, k;
    // for (k = 0; k < n; k++)
    // {
    //     for (i = 0; i < n; i++)
    //     {
    //         for (j = 0; j < n; j++)
    //         {
    //             printf("h_dA: %.2f ", h_dA[k * n * n + i * n + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // for (k = 0; k < n; k++)
    // {
    //     for (i = 0; i < n; i++)
    //     {
    //         for (j = 0; j < n; j++)
    //         {
    //             printf("h_A: %.2f ", h_A[k * n * n + i * n + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }

    if (!checkEqual(h_A, h_dA, n))
    {
        printf("Error: the two matrices are not equal \n");
    }
    double sum = getSum(h_dA, n);
    int total_time_ms = (int)ceil((end_time - start_time) * 1000);
    printf("%lf %d\n", sum, total_time_ms);

    // free memory resources
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_B));
    gpuErrchk(hipHostFree(h_A));
    gpuErrchk(hipHostFree(h_B));
    gpuErrchk(hipDeviceReset());
    return 0;
}